#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cudnnErrCheck(stat) { cudnnErrCheck_((stat), __FILE__, __LINE__); }
void cudnnErrCheck_(hipdnnStatus_t stat, const char *file, int line) {
   if (stat != HIPDNN_STATUS_SUCCESS) {
      fprintf(stderr, "cuDNN Error: %s %s %d\n", hipdnnGetErrorString(stat), file, line);
   }
}

// Global variables 1st LSTMs
int seqLength;

void *x;
void *hx = NULL;
void *cx = NULL;

void *y;
void *hy = NULL;
void *cy = NULL;

void *w;

hipdnnHandle_t cudnnHandle;
hipdnnRNNDescriptor_t rnnDesc;
hipdnnTensorDescriptor_t *xDesc, *yDesc;
hipdnnTensorDescriptor_t hxDesc, cxDesc;
hipdnnTensorDescriptor_t hyDesc, cyDesc;
hipdnnFilterDescriptor_t wDesc;

void *workspace;
size_t workSize;

// Global variables 2nd LSTMs
int seqLength2;

void *x2;
void *hx2 = NULL;
void *cx2 = NULL;

void *y2;
void *hy2 = NULL;
void *cy2 = NULL;

void *w2;

hipdnnHandle_t cudnnHandle2;
hipdnnRNNDescriptor_t rnnDesc2;
hipdnnTensorDescriptor_t *xDesc2, *yDesc2;
hipdnnTensorDescriptor_t hxDesc2, cxDesc2;
hipdnnTensorDescriptor_t hyDesc2, cyDesc2;
hipdnnFilterDescriptor_t wDesc2;

void *workspace2;
size_t workSize2;

void setup_cudnn(int _seqLength, int numLayers, int batch_size, int feature_size) {
    seqLength = _seqLength;
    int hiddenSize = feature_size;
    int inputSize = hiddenSize;
    int miniBatch = batch_size;

    // -------------------------
    // Create cudnn context
    // -------------------------
    cudnnErrCheck(hipdnnCreate(&cudnnHandle));

    // -------------------------
    // Set up inputs and outputs
    // -------------------------
    cudaErrCheck(hipMalloc((void**)&x, seqLength * inputSize * miniBatch * sizeof(DATA_TYPE)));
    cudaErrCheck(hipMalloc((void**)&y, seqLength * hiddenSize * miniBatch * sizeof(DATA_TYPE)));

    xDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));
    yDesc = (hipdnnTensorDescriptor_t*)malloc(seqLength * sizeof(hipdnnTensorDescriptor_t));

    int dimA[3];
    int strideA[3];

    for (int i = 0; i < seqLength; i++) {
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc[i]));
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc[i]));

        dimA[0] = miniBatch;
        dimA[1] = inputSize;
        dimA[2] = 1;

        strideA[0] = dimA[2] * dimA[1];
        strideA[1] = dimA[2];
        strideA[2] = 1;

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc[i], DATA_TYPE_CUDNN, 3, dimA, strideA));

        dimA[0] = miniBatch;
        dimA[1] = hiddenSize;
        dimA[2] = 1;

        strideA[0] = dimA[2] * dimA[1];
        strideA[1] = dimA[2];
        strideA[2] = 1;

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc[i], DATA_TYPE_CUDNN, 3, dimA, strideA));
    }


    dimA[0] = numLayers;
    dimA[1] = miniBatch;
    dimA[2] = hiddenSize;

    strideA[0] = dimA[2] * dimA[1];
    strideA[1] = dimA[2];
    strideA[2] = 1;

    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc));

    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc, DATA_TYPE_CUDNN, 3, dimA, strideA));

    // -------------------------
    // Set up the dropout descriptor (needed for the RNN descriptor)
    // -------------------------

    hipdnnDropoutDescriptor_t dropoutDesc;
    cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    size_t stateSize;
    void *states;
    cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));

    cudaErrCheck(hipMalloc(&states, stateSize));

    cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc,
                cudnnHandle,
                0,
                states,
                stateSize,
                0));

    // -------------------------
    // Set up the RNN descriptor
    // -------------------------
    hipdnnRNNMode_t RNNMode;
    hipdnnRNNAlgo_t RNNAlgo;

    cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc));

    RNNMode = HIPDNN_LSTM;

    RNNAlgo = HIPDNN_RNN_ALGO_STANDARD;

    cudnnErrCheck(hipdnnSetRNNDescriptor_v6(cudnnHandle,
                rnnDesc,
                hiddenSize,
                numLayers,
                dropoutDesc,
                HIPDNN_LINEAR_INPUT,
                HIPDNN_UNIDIRECTIONAL,
                RNNMode,
                RNNAlgo,
                DATA_TYPE_CUDNN));


    // -------------------------
    // Set up parameters
    // -------------------------
    // This needs to be done after the rnn descriptor is set as otherwise
    // we don't know how many parameters we have to allocate

    cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc));

    size_t weightsSize;
    cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc, xDesc[0], &weightsSize, DATA_TYPE_CUDNN));

    int dimW[3];
    dimW[0] = weightsSize / sizeof(DATA_TYPE);
    dimW[1] = 1;
    dimW[2] = 1;

    cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc, DATA_TYPE_CUDNN, HIPDNN_TENSOR_NCHW, 3, dimW));

    cudaErrCheck(hipMalloc((void**)&w,  weightsSize));


    // -------------------------
    // Set up work space and reserved memory
    // -------------------------

    // Need for every pass
    cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc, seqLength, xDesc, &workSize));
    cudaErrCheck(hipMalloc((void**)&workspace, workSize));

}

void setup_cudnn2(int _seqLength, int numLayers, int batch_size, int feature_size) {
    seqLength2 = _seqLength;
    int hiddenSize = feature_size;
    int inputSize = hiddenSize;
    int miniBatch = batch_size;

    // -------------------------
    // Create cudnn context
    // -------------------------
    //cudnnErrCheck(hipdnnCreate(&cudnnHandle2));

    // -------------------------
    // Set up inputs and outputs
    // -------------------------
    cudaErrCheck(hipMalloc((void**)&x2, seqLength2 * inputSize * miniBatch * sizeof(DATA_TYPE)));
    cudaErrCheck(hipMalloc((void**)&y2, seqLength2 * hiddenSize * miniBatch * sizeof(DATA_TYPE)));

    xDesc2 = (hipdnnTensorDescriptor_t*)malloc(seqLength2 * sizeof(hipdnnTensorDescriptor_t));
    yDesc2 = (hipdnnTensorDescriptor_t*)malloc(seqLength2 * sizeof(hipdnnTensorDescriptor_t));

    int dimA[3];
    int strideA[3];

    for (int i = 0; i < seqLength2; i++) {
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&xDesc2[i]));
        cudnnErrCheck(hipdnnCreateTensorDescriptor(&yDesc2[i]));

        dimA[0] = miniBatch;
        dimA[1] = inputSize;
        dimA[2] = 1;

        strideA[0] = dimA[2] * dimA[1];
        strideA[1] = dimA[2];
        strideA[2] = 1;

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(xDesc2[i], DATA_TYPE_CUDNN, 3, dimA, strideA));

        dimA[0] = miniBatch;
        dimA[1] = hiddenSize;
        dimA[2] = 1;

        strideA[0] = dimA[2] * dimA[1];
        strideA[1] = dimA[2];
        strideA[2] = 1;

        cudnnErrCheck(hipdnnSetTensorNdDescriptor(yDesc2[i], DATA_TYPE_CUDNN, 3, dimA, strideA));
    }


    dimA[0] = numLayers;
    dimA[1] = miniBatch;
    dimA[2] = hiddenSize;

    strideA[0] = dimA[2] * dimA[1];
    strideA[1] = dimA[2];
    strideA[2] = 1;

    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hxDesc2));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cxDesc2));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&hyDesc2));
    cudnnErrCheck(hipdnnCreateTensorDescriptor(&cyDesc2));

    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hxDesc2, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cxDesc2, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(hyDesc2, DATA_TYPE_CUDNN, 3, dimA, strideA));
    cudnnErrCheck(hipdnnSetTensorNdDescriptor(cyDesc2, DATA_TYPE_CUDNN, 3, dimA, strideA));

    // -------------------------
    // Set up the dropout descriptor (needed for the RNN descriptor)
    // -------------------------

    hipdnnDropoutDescriptor_t dropoutDesc;
    cudnnErrCheck(hipdnnCreateDropoutDescriptor(&dropoutDesc));

    size_t stateSize;
    void *states;
    cudnnErrCheck(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));

    cudaErrCheck(hipMalloc(&states, stateSize));

    cudnnErrCheck(hipdnnSetDropoutDescriptor(dropoutDesc,
                cudnnHandle,
                0,
                states,
                stateSize,
                0));

    // -------------------------
    // Set up the RNN descriptor
    // -------------------------
    hipdnnRNNMode_t RNNMode;
    hipdnnRNNAlgo_t RNNAlgo;

    cudnnErrCheck(hipdnnCreateRNNDescriptor(&rnnDesc2));

    RNNMode = HIPDNN_LSTM;

    RNNAlgo = HIPDNN_RNN_ALGO_STANDARD;

    cudnnErrCheck(hipdnnSetRNNDescriptor_v6(cudnnHandle,
                rnnDesc2,
                hiddenSize,
                numLayers,
                dropoutDesc,
                HIPDNN_LINEAR_INPUT,
                HIPDNN_UNIDIRECTIONAL,
                RNNMode,
                RNNAlgo,
                DATA_TYPE_CUDNN));


    // -------------------------
    // Set up parameters
    // -------------------------
    // This needs to be done after the rnn descriptor is set as otherwise
    // we don't know how many parameters we have to allocate

    cudnnErrCheck(hipdnnCreateFilterDescriptor(&wDesc2));

    size_t weightsSize;
    cudnnErrCheck(hipdnnGetRNNParamsSize(cudnnHandle, rnnDesc2, xDesc2[0], &weightsSize, DATA_TYPE_CUDNN));

    int dimW[3];
    dimW[0] = weightsSize / sizeof(DATA_TYPE);
    dimW[1] = 1;
    dimW[2] = 1;

    cudnnErrCheck(hipdnnSetFilterNdDescriptor(wDesc2, DATA_TYPE_CUDNN, HIPDNN_TENSOR_NCHW, 3, dimW));

    cudaErrCheck(hipMalloc((void**)&w2,  weightsSize));


    // -------------------------
    // Set up work space and reserved memory
    // -------------------------

    // Need for every pass
    cudnnErrCheck(hipdnnGetRNNWorkspaceSize(cudnnHandle, rnnDesc2, seqLength2, xDesc2, &workSize2));
    cudaErrCheck(hipMalloc((void**)&workspace2, workSize2));

}

float run_cudnn(DATA_TYPE *raw_Weights, DATA_TYPE *raw_biases, DATA_TYPE *raw_x, DATA_TYPE *raw_y) {
    cudaErrCheck(hipDeviceSynchronize());

    // Initialise inputs
    hipMemcpy(x, raw_x, FEATURE_SIZE * BATCH_SIZE * SEQ_LENGTH * sizeof(DATA_TYPE),
               hipMemcpyKind::hipMemcpyHostToDevice);

    for (int layer = 0; layer < NUM_LAYERS; layer++) {
        for (int linLayerID = 0; linLayerID < 8; linLayerID++) {
            hipdnnFilterDescriptor_t linLayerMatDesc;
            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
            DATA_TYPE *linLayerMat;

            cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams(cudnnHandle,
                        rnnDesc,
                        layer,
                        xDesc[0],
                        wDesc,
                        w,
                        linLayerID,
                        linLayerMatDesc,
                        (void**)&linLayerMat));

            hipdnnDataType_t dataType;
            hipdnnTensorFormat_t format;
            int nbDims;
            int filterDimA[3];
            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc,
                        3,
                        &dataType,
                        &format,
                        &nbDims,
                        filterDimA));

            hipMemcpy(linLayerMat, raw_Weights + FEATURE_SIZE * FEATURE_SIZE * (linLayerID + 8 * layer),
                       FEATURE_SIZE * FEATURE_SIZE * sizeof(DATA_TYPE), hipMemcpyKind::hipMemcpyHostToDevice);

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));

            hipdnnFilterDescriptor_t linLayerBiasDesc;
            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
            DATA_TYPE *linLayerBias;

            cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams(cudnnHandle,
                        rnnDesc,
                        layer,
                        xDesc[0],
                        wDesc,
                        w,
                        linLayerID,
                        linLayerBiasDesc,
                        (void**)&linLayerBias));

            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc,
                        3,
                        &dataType,
                        &format,
                        &nbDims,
                        filterDimA));

            // We use merged bias
            if (linLayerID < 4) {
                hipMemcpy(linLayerBias,
                           raw_biases + FEATURE_SIZE * (linLayerID + 4 * layer),
                           FEATURE_SIZE * sizeof(DATA_TYPE),
                           hipMemcpyKind::hipMemcpyHostToDevice);
            } else {
                hipMemset(linLayerBias, 0, FEATURE_SIZE * sizeof(DATA_TYPE));
            }

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
        }
    }

    cudaErrCheck(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float timeForward;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));

    cudaErrCheck(hipEventRecord(start));

    cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle,
                rnnDesc,
                seqLength,
                xDesc,
                x,
                hxDesc,
                hx,
                cxDesc,
                cx,
                wDesc,
                w,
                yDesc,
                y,
                hyDesc,
                hy,
                cyDesc,
                cy,
                workspace,
                workSize));

    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&timeForward, start, stop));

    // Make double-sure everything is finished before we copy for result checking.
    hipDeviceSynchronize();

    hipMemcpy(raw_y, y, FEATURE_SIZE * BATCH_SIZE * SEQ_LENGTH * sizeof(DATA_TYPE),
               hipMemcpyKind::hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return timeForward;
}

void free_cudnn() {
    hipFree(x);
    hipFree(y);
    hipFree(workspace);
    hipFree(w);

    hipdnnDestroy(cudnnHandle);
}

float run_cudnn2(DATA_TYPE *raw_Weights, DATA_TYPE *raw_biases, DATA_TYPE *raw_x, DATA_TYPE *raw_y) {
    cudaErrCheck(hipDeviceSynchronize());

    // Initialise inputs

    hipMemcpy(raw_x, x2, FEATURE_SIZE * BATCH_SIZE * (SEQ_LENGTH / 2) * sizeof(DATA_TYPE),
               hipMemcpyKind::hipMemcpyDeviceToHost);

    for (int layer = 0; layer < NUM_LAYERS; layer++) {
        for (int linLayerID = 0; linLayerID < 8; linLayerID++) {
            hipdnnFilterDescriptor_t linLayerMatDesc;
            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerMatDesc));
            DATA_TYPE *linLayerMat;

            cudnnErrCheck(hipdnnGetRNNLinLayerMatrixParams(cudnnHandle,
                        rnnDesc2,
                        layer,
                        xDesc2[0],
                        wDesc2,
                        w2,
                        linLayerID,
                        linLayerMatDesc,
                        (void**)&linLayerMat));

            hipdnnDataType_t dataType;
            hipdnnTensorFormat_t format;
            int nbDims;
            int filterDimA[3];
            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerMatDesc,
                        3,
                        &dataType,
                        &format,
                        &nbDims,
                        filterDimA));

            hipMemcpy(linLayerMat, raw_Weights + FEATURE_SIZE * FEATURE_SIZE * (linLayerID + 8 * layer),
                       FEATURE_SIZE * FEATURE_SIZE * sizeof(DATA_TYPE), hipMemcpyKind::hipMemcpyHostToDevice);

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerMatDesc));

            hipdnnFilterDescriptor_t linLayerBiasDesc;
            cudnnErrCheck(hipdnnCreateFilterDescriptor(&linLayerBiasDesc));
            DATA_TYPE *linLayerBias;

            cudnnErrCheck(hipdnnGetRNNLinLayerBiasParams(cudnnHandle,
                        rnnDesc2,
                        layer,
                        xDesc2[0],
                        wDesc2,
                        w2,
                        linLayerID,
                        linLayerBiasDesc,
                        (void**)&linLayerBias));

            cudnnErrCheck(hipdnnGetFilterNdDescriptor(linLayerBiasDesc,
                        3,
                        &dataType,
                        &format,
                        &nbDims,
                        filterDimA));

            // We use merged bias
            if (linLayerID < 4) {
                hipMemcpy(linLayerBias,
                           raw_biases + FEATURE_SIZE * (linLayerID + 4 * layer),
                           FEATURE_SIZE * sizeof(DATA_TYPE),
                           hipMemcpyKind::hipMemcpyHostToDevice);
            } else {
                hipMemset(linLayerBias, 0, FEATURE_SIZE * sizeof(DATA_TYPE));
            }

            cudnnErrCheck(hipdnnDestroyFilterDescriptor(linLayerBiasDesc));
        }
    }

    cudaErrCheck(hipDeviceSynchronize());

    hipEvent_t start, stop;
    float timeForward;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));

    cudaErrCheck(hipEventRecord(start));

    cudnnErrCheck(hipdnnRNNForwardInference(cudnnHandle,
                rnnDesc2,
                seqLength2,
                xDesc2,
                x2,
                hxDesc2,
                hx2,
                cxDesc2,
                cx2,
                wDesc2,
                w2,
                yDesc2,
                y2,
                hyDesc2,
                hy2,
                cyDesc2,
                cy2,
                workspace2,
                workSize2));

    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&timeForward, start, stop));

    // Make double-sure everything is finished before we copy for result checking.
    hipDeviceSynchronize();

    hipMemcpy(raw_y, y2, FEATURE_SIZE * BATCH_SIZE * (SEQ_LENGTH/2) * sizeof(DATA_TYPE),
               hipMemcpyKind::hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return timeForward;
}

void free_cudnn2() {
    hipFree(x2);
    hipFree(y2);
    hipFree(workspace2);
    hipFree(w2);

    //hipdnnDestroy(cudnnHandle2);
}

static __global__ void _kernel_maxpool(DATA_TYPE *src, DATA_TYPE *dst)
{
  int batch = blockIdx.x;
  int feature = threadIdx.x;
  for (int s = 0; s< SEQ_LENGTH/2; s++){
    dst[s * BATCH_SIZE * FEATURE_SIZE + batch * FEATURE_SIZE + feature] = max(
                                                                                src[(2 * s) * BATCH_SIZE * FEATURE_SIZE + batch * FEATURE_SIZE + feature],
                                                                                src[(2 * s + 1) * BATCH_SIZE * FEATURE_SIZE + batch * FEATURE_SIZE + feature]
                                                                              );
  }
}

void downsampling_maxpool_wrapper(){
  _kernel_maxpool<<<BATCH_SIZE, FEATURE_SIZE>>>((float *) y, (float *)x2);
  // SYNCHRONIZE
  hipDeviceSynchronize();
}
